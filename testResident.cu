#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int maxResidentThreads = prop.multiProcessorCount * prop.maxThreadsPerMultiProcessor;
    printf("Max thread residenti simultaneamente: %d\n", maxResidentThreads);
    return 0;
}
